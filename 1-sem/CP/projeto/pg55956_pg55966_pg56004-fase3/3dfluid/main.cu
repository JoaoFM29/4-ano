#include "hip/hip_runtime.h"
#include "EventManager.h"
#include "fluid_solver.h"
#include <iostream>
#include <vector>

#define SIZE 168

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))

// Globals for the grid size
static int M = SIZE;
static int N = SIZE;
static int O = SIZE;
static float dt = 0.1f;      // Time delta
static float diff = 0.0001f; // Diffusion constant
static float visc = 0.0001f; // Viscosity constant

// Fluid simulation arrays
static float *dens;

// Fluid simulation on cuda

static float *d_u, *d_v, *d_w, *d_u_prev, *d_v_prev, *d_w_prev;
static float *d_dens, *d_dens_prev;

// Function to allocate simulation data
int allocate_data() {
  int size = (M + 2) * (N + 2) * (O + 2);
  dens = new float[size];
  hipMalloc((void **)&d_u, size * sizeof(float));
  hipMalloc((void **)&d_v, size * sizeof(float));
  hipMalloc((void **)&d_w, size * sizeof(float));
  hipMalloc((void **)&d_u_prev, size * sizeof(float));
  hipMalloc((void **)&d_v_prev, size * sizeof(float));
  hipMalloc((void **)&d_w_prev, size * sizeof(float));
  hipMalloc((void **)&d_dens, size * sizeof(float));
  hipMalloc((void **)&d_dens_prev, size * sizeof(float));
  
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA Error after Malloc: %s\n", hipGetErrorString(err));
      return 0;
  }

  if (!dens) {
    std::cerr << "Cannot allocate memory" << std::endl;
    return 0;
  }
  return 1;
}

// Function to clear the data (set all to zero)
void clear_data() {
  int size = (M + 2) * (N + 2) * (O + 2);

  hipMemset(d_dens, 0, size);
  hipMemset(d_dens_prev, 0, size);
  hipMemset(d_u, 0, size);
  hipMemset(d_v, 0, size);
  hipMemset(d_w, 0, size);
  hipMemset(d_u_prev, 0, size);
  hipMemset(d_v_prev, 0, size);
  hipMemset(d_w_prev, 0, size);

}

// Free allocated memory
void free_data() {

  delete[] dens;
  hipFree(d_u);
  hipFree(d_v);
  hipFree(d_w);
  hipFree(d_u_prev);
  hipFree(d_v_prev);
  hipFree(d_w_prev);
  hipFree(d_dens);
  hipFree(d_dens_prev);
}


__global__
void apply_events_kernel(int M,int N,int O,float *u,float *v,float *w,float *dens,Event event){


  int i = M / 2, j = N / 2, k = O / 2; // Assume all events affect the center
    int idx = IX(i, j, k);

        if (event.type == ADD_SOURCE) {
            // Add density source
            dens[idx] = event.density;
        } else if (event.type == APPLY_FORCE) {
            // Apply force vector
            u[idx] = event.force.x;
            v[idx] = event.force.y;
            w[idx] = event.force.z;
        }
    

}
// Apply events (source or force) for the current timestep
void apply_events(const std::vector<Event> &events) {


  for (const auto &event : events) {
    dim3 threadsPerBlock(32);
    dim3 numBlocks(1); 
    apply_events_kernel<<<numBlocks, threadsPerBlock>>>(M,N,O,d_u, d_v, d_w, d_dens, event);
  }

}

/*
__global__ void sum_density_kernel(float* d_dens, float* d_partial_sums, int size) {
    extern __shared__ float sdata[];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int local_tid = threadIdx.x;

    // Load data into shared memory
    sdata[local_tid] = (tid < size) ? d_dens[tid] : 0.0f;
    __syncthreads();

    // Perform reduction within the block
    for (int stride = blockDim.x / 2; stride > 0; stride /= 2) {
        if (local_tid < stride) {
            sdata[local_tid] += sdata[local_tid + stride];
        }
        __syncthreads();
    }

    // Write block's partial sum to global memory
    if (local_tid == 0) {
        d_partial_sums[blockIdx.x] = sdata[0];
    }
}


float sum_density() {
    int size = (M + 2) * (N + 2) * (O + 2);
    int threads_per_block = 256;
    int num_blocks = (size + threads_per_block - 1) / threads_per_block;

    // Allocate memory for partial sums on the GPU
    float* d_partial_sums;
    hipMalloc(&d_partial_sums, num_blocks * sizeof(float));

    // Launch reduction kernel
    sum_density_kernel<<<num_blocks, threads_per_block, threads_per_block * sizeof(float)>>>(d_dens, d_partial_sums, size);

    // Copy partial sums back to the host
    std::vector<float> partial_sums(num_blocks);
    hipMemcpy(partial_sums.data(), d_partial_sums, num_blocks * sizeof(float), hipMemcpyDeviceToHost);

    // Perform final summation on the host
    float total_density = 0.0f;
    for (float partial : partial_sums) {
        total_density += partial;
    }

    // Free GPU memory for partial sums
    hipFree(d_partial_sums);

    return total_density;
}
*/

float sum_density() {

  float total_density = 0.0f;
  int size = (M + 2) * (N + 2) * (O + 2);
  hipMemcpy(dens,d_dens, sizeof(float) * size, hipMemcpyDeviceToHost);
  for (int i = 0; i < size; i++) {
    total_density += dens[i];
  }
  return total_density;
}

// Simulation loop
void simulate(EventManager &eventManager, int timesteps) {
  for (int t = 0; t < timesteps; t++) {
    // Get the events for the current timestep
    std::vector<Event> events = eventManager.get_events_at_timestamp(t);

    // Apply events to the simulation
    apply_events(events);

    // Perform the simulation steps
    vel_step(M, N, O, d_u, d_v, d_w, d_u_prev, d_v_prev, d_w_prev, visc, dt);
    dens_step(M, N, O,d_dens, d_dens_prev, d_u, d_v, d_w, diff, dt);
  }
}

int main() {
  // Initialize EventManager
  EventManager eventManager;
  eventManager.read_events("events.txt");

  // Get the total number of timesteps from the event file
  int timesteps = eventManager.get_total_timesteps();

  // Allocate and clear data
  if (!allocate_data())
    return -1;
  clear_data();

  // Run simulation with events
  simulate(eventManager, timesteps);

  // Print total density at the end of simulation
  float total_density = sum_density();
  std::cout << "Total density after " << timesteps
            << " timesteps: " << total_density << std::endl;

  // Free memory
  free_data();

  return 0;
}