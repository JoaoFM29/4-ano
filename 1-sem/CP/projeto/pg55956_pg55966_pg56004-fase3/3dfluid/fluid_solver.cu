#include "hip/hip_runtime.h"
#include "fluid_solver.h"
#include <cmath>
#include <iostream>
#include <omp.h>

#define IX(i, j, k) ((i) + (M + 2) * (j) + (M + 2) * (N + 2) * (k))
#define SWAP(x0, x)                                                            \
  {                                                                            \
    float *tmp = x0;                                                           \
    x0 = x;                                                                    \
    x = tmp;                                                                   \
  }
#define MAX(a, b) (((a) > (b)) ? (a) : (b))
#define LINEARSOLVERTIMES 20


__global__ void add_source_kernel(int size, float *x, const float *s, float dt) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        x[idx] += dt * s[idx];
    }
}

// Add sources (density or velocity)
void add_source(int M, int N, int O, float *x, const float *s, float dt) {
    int size = ((M + 2) * (N + 2) * (O + 2));

    // Configure kernel launch parameters
    int threadsPerBlock = 256;
    int blocksPerGrid = (( size ) + threadsPerBlock - 1) / threadsPerBlock;

    // Launch the kernel
    add_source_kernel<<<blocksPerGrid, threadsPerBlock>>>(size,x,s, dt);
}

// Kernel for handling the faces of the cube
__global__
void set_bnd_faces_kernel_a(int M, int N, int O, int b, float* x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; // Skip boundary (start at 1)
    int idy = blockIdx.y * blockDim.y + threadIdx.y + 1; // Skip boundary (start at 1)

    if (idx <= M && idy <= N) {
        x[IX(idx, idy, 0)] = b == 3 ? -x[IX(idx, idy, 1)] : x[IX(idx, idy, 1)];
        x[IX(idx, idy, O + 1)] = b == 3 ? -x[IX(idx, idy, O)] : x[IX(idx, idy, O)];
    }
}

__global__
void set_bnd_faces_kernel_b(int M, int N, int O, int b, float* x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; // Skip boundary (start at 1)
    int idy = blockIdx.y * blockDim.y + threadIdx.y + 1; // Skip boundary (start at 1)

    if (idx <= N && idy <= O) {
        x[IX(0, idx, idy)] = b == 1 ? -x[IX(1, idx, idy)] : x[IX(1, idx, idy)];
        x[IX(M + 1, idx, idy)] = b == 1 ? -x[IX(M, idx, idy)] : x[IX(M, idx, idy)];
    }
}

__global__
void set_bnd_faces_kernel_c(int M, int N, int O, int b, float* x) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x + 1; // Skip boundary (start at 1)
    int idy = blockIdx.y * blockDim.y + threadIdx.y + 1; // Skip boundary (start at 1)

    if (idx <= M && idy <= O) {
        x[IX(idx, 0, idy)] = b == 2 ? -x[IX(idx, 1, idy)] : x[IX(idx, 1, idy)];
        x[IX(idx, N + 1, idy)] = b == 2 ? -x[IX(idx, N, idy)] : x[IX(idx, N, idy)];
    }
}

void launch_set_bnd_faces_kernels(int M, int N, int O, int b, float* d_x) {
    // Define block and grid dimensions for all kernels
    dim3 blockDim(16, 16); // Adjust as necessary based on GPU resources

    dim3 gridDim_a((M + 2+ blockDim.x - 1) / blockDim.x, (N + 2 + blockDim.y - 1) / blockDim.y);
    set_bnd_faces_kernel_a<<<gridDim_a, blockDim>>>(M, N, O, b, d_x);


    dim3 gridDim_b((N + 2 + blockDim.x - 1) / blockDim.x, (O  + 2+ blockDim.y - 1) / blockDim.y);
    set_bnd_faces_kernel_b<<<gridDim_b, blockDim>>>(M, N, O, b, d_x);


    dim3 gridDim_c((M + 2 + blockDim.x - 1) / blockDim.x, (O + 2 + blockDim.y - 1) / blockDim.y);
    set_bnd_faces_kernel_c<<<gridDim_c, blockDim>>>(M, N, O, b, d_x);

}
// Kernel for handling the corners
__global__ void set_bnd_corners_kernel(int M, int N, int O, float* x) {

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx == 0) {
        x[IX(0, 0, 0)] = 0.33f * (x[IX(1, 0, 0)] + x[IX(0, 1, 0)] + x[IX(0, 0, 1)]);
        x[IX(M + 1, 0, 0)] = 0.33f * (x[IX(M, 0, 0)] + x[IX(M + 1, 1, 0)] + x[IX(M + 1, 0, 1)]);
        x[IX(0, N + 1, 0)] = 0.33f * (x[IX(1, N + 1, 0)] + x[IX(0, N, 0)] + x[IX(0, N + 1, 1)]);
        x[IX(M + 1, N + 1, 0)] = 0.33f * (x[IX(M, N + 1, 0)] + x[IX(M + 1, N, 0)] + x[IX(M + 1, N + 1, 1)]);
    }
}
// Main function to set boundary conditions
void set_bnd_cuda(int M, int N, int O, int b, float* d_x) {
   
    launch_set_bnd_faces_kernels(M,N,O,b,d_x);
    // Configuration for corners kernel (single block with single thread is sufficient)
    dim3 cornerThreads(32, 1, 1);  
    dim3 cornerBlocks(1, 1, 1);
  
    set_bnd_corners_kernel<<<cornerBlocks, cornerThreads>>>(M, N, O, d_x);
}

__device__ int getGlobalBlockId() {
    return blockIdx.x + 
           blockIdx.y * gridDim.x + 
           blockIdx.z * gridDim.x * gridDim.y;
}

template<int BLOCK_SIZE>
__global__ void red_phase_kernel(int M, int N, int O, float *x, float *x0, float a, float c, float *max_change) {
    extern __shared__ float block_changes[];  // Dynamic shared memory

    // Calculate linear thread ID
    int lid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    
    // Calculate 3D indices
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    
    // Compute change
    float change = 0.0f;
    if (i <= M && j <= N && k <= O && (i + j + k) % 2 == 0) {
        int idx = IX(i, j, k);
        float old_x = x[idx];
        x[idx] = (x0[idx] + 
                    a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                        x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                        x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) / c;
        change = fabs(x[idx] - old_x);
    }
    
    // Store in shared memory
    block_changes[lid] = change;
    __syncthreads();
    
    // Reduction optimized for 1024 threads
    if ( BLOCK_SIZE >= 1024) {
        if (lid < 512) {
            block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 512]);
        }
        __syncthreads();
    }

    if ( BLOCK_SIZE >= 512) {
    if (lid < 256) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 256]);
    }
    __syncthreads();
     }
    if ( BLOCK_SIZE >= 256) {
    if (lid < 128) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 128]);
    }
    __syncthreads();
     }
     if ( BLOCK_SIZE >= 128) {
    if (lid < 64) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 64]);
    }
    __syncthreads();
    }    
    // Last 64 elements
    if (lid < 32) {
        volatile float* smem = block_changes;
        smem[lid] = fmaxf(smem[lid], smem[lid + 32]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 16]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 8]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 4]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 2]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 1]);
    }
    __syncthreads();
    // Write result
    if (lid == 0)  max_change[getGlobalBlockId()] = block_changes[0];
    
}

template<int BLOCK_SIZE>
__global__ void black_phase_kernel(int M, int N, int O, float *x, float *x0, float a, float c, float *max_change) {
    extern __shared__ float block_changes[];  // Dynamic shared memory

    // Calculate linear thread ID
    int lid = threadIdx.x + threadIdx.y * blockDim.x + threadIdx.z * blockDim.x * blockDim.y;
    
    // Calculate 3D indices
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    
    // Compute change
    float change = 0.0f;
    if (i <= M && j <= N && k <= O && (i + j + k) % 2 == 1) {
        int idx = IX(i, j, k);
        float old_x = x[idx];
        x[idx] = (x0[idx] + 
                    a * (x[IX(i - 1, j, k)] + x[IX(i + 1, j, k)] +
                        x[IX(i, j - 1, k)] + x[IX(i, j + 1, k)] +
                        x[IX(i, j, k - 1)] + x[IX(i, j, k + 1)])) / c;
        change = fabs(x[idx] - old_x);
    }
    
    // Store in shared memory
    block_changes[lid] = change;
    __syncthreads();
    
  // Reduction optimized for 1024 threads
 if ( BLOCK_SIZE >= 1024) {
    if (lid < 512) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 512]);
    }
    __syncthreads();
 }

    if ( BLOCK_SIZE >= 512) {
    if (lid < 256) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 256]);
    }
    __syncthreads();
     }
    if ( BLOCK_SIZE >= 256) {
    if (lid < 128) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 128]);
    }
    __syncthreads();
     }
     if ( BLOCK_SIZE >= 128) {
    if (lid < 64) {
        block_changes[lid] = fmaxf(block_changes[lid], block_changes[lid + 64]);
    }
    __syncthreads();
    }    
    // Last 64 elements
    if (lid < 32) {
        volatile float* smem = block_changes;
        smem[lid] = fmaxf(smem[lid], smem[lid + 32]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 16]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 8]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 4]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 2]);
        smem[lid] = fmaxf(smem[lid], smem[lid + 1]);
    }
    __syncthreads();

    if (lid == 0) max_change[getGlobalBlockId()] = block_changes[0];
}

template<int BLOCK_SIZE>
__global__ void reduceMaxKernel(float* input, float* output, int size) {
    extern __shared__ float sdata[];

    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * (2 * blockDim.x) + threadIdx.x;

    // Load data into shared memory
    sdata[tid] = (i < size) ? input[i] : -INFINITY;
    if (i + blockDim.x < size) {
        sdata[tid] = fmaxf(sdata[tid], input[i + blockDim.x]);
    }
    __syncthreads();

     if ( BLOCK_SIZE >= 1024) {
    if (tid < 512) {
         sdata[tid] = fmaxf( sdata[tid],  sdata[tid + 512]);
    }
    __syncthreads();
 }

    if ( BLOCK_SIZE >= 512) {
    if (tid < 256) {
         sdata[tid ] = fmaxf( sdata[tid ],  sdata[tid + 256]);
    }
    __syncthreads();
     }
    if ( BLOCK_SIZE >= 256) {
    if (tid  < 128) {
         sdata[tid ] = fmaxf( sdata[tid],  sdata[tid + 128]);
    }
    __syncthreads();
     }
     if ( BLOCK_SIZE >= 128) {
    if (tid < 64) {
         sdata[tid] = fmaxf( sdata[tid],  sdata[tid + 64]);
    }
    __syncthreads();
    }    
    // Last 64 elements
    if (tid < 32) {
        volatile float* smem =  sdata;
        smem[tid] = fmaxf(smem[tid], smem[tid + 32]);
        smem[tid] = fmaxf(smem[tid], smem[tid + 16]);
        smem[tid] = fmaxf(smem[tid], smem[tid + 8]);
        smem[tid] = fmaxf(smem[tid], smem[tid + 4]);
        smem[tid] = fmaxf(smem[tid], smem[tid + 2]);
        smem[tid] = fmaxf(smem[tid], smem[tid + 1]);
    }
    __syncthreads();
    // Write the maximum for this block to global memory
    if (tid == 0) output[blockIdx.x] = sdata[0];
}

__global__ void compareMaxChangesKernel(float* redChanges, float* blackChanges, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        redChanges[idx] = fmaxf(redChanges[idx], blackChanges[idx]);
    }
}

int getTotalBlocks(const dim3& numBlocks) {
    return numBlocks.x * numBlocks.y * numBlocks.z;
}

int getTotalThreadsPerBlock(const dim3& threadsPerBlock){
    return threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z;
}

void lin_solve(int M, int N, int O, int b, float *x, float *x0, float a, float c) {
    float tol = 1e-7;
    float *d_max_change_red,*d_max_change_black, *d_final_max;
    float max_change;
    
    dim3 threadsPerBlock(16, 16, 1);
    dim3 numBlocks(
        (M  + 15) / 16,
        (N + 15) / 16,
        (O)
    );
    int totalBlocks = getTotalBlocks(numBlocks);
    int l = 0;
    
    hipMalloc((void**) &d_max_change_red, sizeof(float) * totalBlocks);
    hipMalloc((void**) &d_max_change_black, sizeof(float) * totalBlocks);
    hipMalloc((void**) &d_final_max, sizeof(float));

    const int REDUCE_THREADS = 256;
    int reduceBlocks = (totalBlocks + REDUCE_THREADS - 1) / REDUCE_THREADS;
    const int tt = 256;

    do {
        max_change = 0.0f;

        red_phase_kernel<tt><<<numBlocks, threadsPerBlock,tt * sizeof(float)>>>(M, N, O, x, x0, a, c, d_max_change_red);

        black_phase_kernel<tt><<<numBlocks, threadsPerBlock,tt * sizeof(float)>>>(M, N, O, x, x0, a, c, d_max_change_black);

        compareMaxChangesKernel<<<reduceBlocks, REDUCE_THREADS>>>(d_max_change_red, d_max_change_black, totalBlocks);

        reduceMaxKernel<REDUCE_THREADS><<<reduceBlocks, REDUCE_THREADS, REDUCE_THREADS * sizeof(float)>>>
            (d_max_change_red, d_final_max, totalBlocks);

        int currentBlocks = reduceBlocks;
        while (currentBlocks > 1) {
            int nextBlocks = (currentBlocks + REDUCE_THREADS - 1) / REDUCE_THREADS;
            reduceMaxKernel<REDUCE_THREADS><<<nextBlocks, REDUCE_THREADS, REDUCE_THREADS * sizeof(float)>>>
                (d_final_max, d_final_max, currentBlocks);

            currentBlocks = nextBlocks;
        }
        
        set_bnd_cuda(M, N, O, b, x);
        hipDeviceSynchronize();
        hipMemcpy(&max_change, d_final_max, sizeof(float), hipMemcpyDeviceToHost);
    
    } while (max_change > tol && ++l < 20);

    hipFree(d_max_change_red);
    hipFree(d_max_change_black);
    hipFree(d_final_max);
}
// Diffusion step (uses implicit method)
void diffuse(int M, int N, int O, int b, float *x, float *x0, float diff,
             float dt) {
  int max = MAX(MAX(M, N), O);
  float a = dt * diff * max * max;
  lin_solve(M, N, O, b, x, x0, a, 1 + 6 * a);
}

__global__ void advect_kernel(int M, int N, int O, int b, float *d, float *d0, float *u, float *v,
            float *w, float dt) {

  float dtX = dt * M, dtY = dt * N, dtZ = dt * O;
  int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
  int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
  int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
  if ( k > O || j > N || i > M) return;


        float x = i - dtX * u[IX(i, j, k)];
        float y = j - dtY * v[IX(i, j, k)];
        float z = k - dtZ * w[IX(i, j, k)];

        // Clamp to grid boundaries
        if (x < 0.5f)
          x = 0.5f;
        if (x > M + 0.5f)
          x = M + 0.5f;
        if (y < 0.5f)
          y = 0.5f;
        if (y > N + 0.5f)
          y = N + 0.5f;
        if (z < 0.5f)
          z = 0.5f;
        if (z > O + 0.5f)
          z = O + 0.5f;

        int i0 = (int)x, i1 = i0 + 1;
        int j0 = (int)y, j1 = j0 + 1;
        int k0 = (int)z, k1 = k0 + 1;

        float s1 = x - i0, s0 = 1 - s1;
        float t1 = y - j0, t0 = 1 - t1;
        float u1 = z - k0, u0 = 1 - u1;

        d[IX(i, j, k)] =
            s0 * (t0 * (u0 * d0[IX(i0, j0, k0)] + u1 * d0[IX(i0, j0, k1)]) +
                  t1 * (u0 * d0[IX(i0, j1, k0)] + u1 * d0[IX(i0, j1, k1)])) +
            s1 * (t0 * (u0 * d0[IX(i1, j0, k0)] + u1 * d0[IX(i1, j0, k1)]) +
                  t1 * (u0 * d0[IX(i1, j1, k0)] + u1 * d0[IX(i1, j1, k1)]));
 
}

// Advection step (uses velocity field to move quantities)
void advect(int M, int N, int O, int b, float *d, float *d0, float *u, float *v,
            float *w, float dt) {
  
    // Configure kernel launch parameters
    dim3 threadsPerBlock(16, 16, 1);  // 256 threads per block
    dim3 numBlocks(
        (M + 15 ) / 16,
        (N + 15 ) / 16,
        (O)
    );

    // Launch the kernel
    advect_kernel<<<numBlocks, threadsPerBlock>>>(M, N, O,b,d,d0,u,v,w, dt);
    set_bnd_cuda(M, N, O, b, d);


}

// Projection step to ensure incompressibility (make the velocity field
// divergence-free)
__global__ void project_kernel_a(int M, int N, int O, float *u, float *v, float *w, float *p,
             float *div) {
    
    int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
    int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
    int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
    if ( k > O || j > N || i > M) return;

    div[IX(i, j, k)] =
        -0.5f *
        (u[IX(i + 1, j, k)] - u[IX(i - 1, j, k)] + v[IX(i, j + 1, k)] -
          v[IX(i, j - 1, k)] + w[IX(i, j, k + 1)] - w[IX(i, j, k - 1)]) /
        MAX(M, MAX(N, O));
    p[IX(i, j, k)] = 0;

}

__global__ void project_kernel_b(int M, int N, int O, float *u, float *v, float *w, float *p,
             float *div) {

        int i = blockIdx.x * blockDim.x + threadIdx.x + 1;
        int j = blockIdx.y * blockDim.y + threadIdx.y + 1;
        int k = blockIdx.z * blockDim.z + threadIdx.z + 1;
        if ( k > O || j > N || i > M) return;

        u[IX(i, j, k)] -= 0.5f * (p[IX(i + 1, j, k)] - p[IX(i - 1, j, k)]);
        v[IX(i, j, k)] -= 0.5f * (p[IX(i, j + 1, k)] - p[IX(i, j - 1, k)]);
        w[IX(i, j, k)] -= 0.5f * (p[IX(i, j, k + 1)] - p[IX(i, j, k - 1)]);
}

void project(int M, int N, int O, float *u, float *v, float *w, float *p,
             float *div) {

  // Configure kernel launch parameters
    dim3 threadsPerBlock(16, 16, 1);  // 512 threads per block
    dim3 numBlocks(
        (M + 15 ) / 16,
        (N + 15  ) / 16,
        (O)
    );

    // Launch the kernel
    project_kernel_a<<<numBlocks, threadsPerBlock>>>(M, N, O,u,v,w,p,div);


    set_bnd_cuda(M, N, O, 0,div);
    set_bnd_cuda(M, N, O, 0,p);

    lin_solve(M, N, O, 0, p, div, 1, 6);


    project_kernel_b<<<numBlocks, threadsPerBlock>>>(M, N, O,u,v,w,p,div);
      
    set_bnd_cuda(M, N, O, 1,u);
    set_bnd_cuda(M, N, O, 2,v);
    set_bnd_cuda(M, N, O, 3,w);

}

// Step function for density
void dens_step(int M, int N, int O, float *x, float *x0, float *u, float *v,
               float *w, float diff, float dt) {
  add_source(M, N, O, x, x0, dt);
  SWAP(x0, x);
  diffuse(M, N, O, 0, x, x0, diff, dt);
  SWAP(x0, x);
  advect(M, N, O, 0, x, x0, u, v, w, dt);
}

// Step function for velocity
void vel_step(int M, int N, int O, float *u, float *v, float *w, float *u0,
              float *v0, float *w0, float visc, float dt) {
  add_source(M, N, O, u, u0, dt);
  add_source(M, N, O, v, v0, dt);
  add_source(M, N, O, w, w0, dt);
  SWAP(u0, u);
  diffuse(M, N, O, 1, u, u0, visc, dt);
  SWAP(v0, v);
  diffuse(M, N, O, 2, v, v0, visc, dt);
  SWAP(w0, w);
  diffuse(M, N, O, 3, w, w0, visc, dt);
  project(M, N, O, u, v, w, u0, v0);
  SWAP(u0, u);
  SWAP(v0, v);
  SWAP(w0, w);
  advect(M, N, O, 1, u, u0, u0, v0, w0, dt);
  advect(M, N, O, 2, v, v0, u0, v0, w0, dt);
  advect(M, N, O, 3, w, w0, u0, v0, w0, dt);
  project(M, N, O, u, v, w, u0, v0);
}